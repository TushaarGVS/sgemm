/**
 * @file runner.cu
 * @brief Runner for the SGEMM kernels.
 * @copyright Copyright (C) 2025, Tushaar Gangavarapu <TG352@cornell.edu>.
 * @date 2025-06-25
 */

#include "kernels/1_sgemm_naive.cuh"
#include "kernels/2_sgemm_gmem_coalesce.cuh"
#include "kernels/3_sgemm_smem_tiling.cuh"
#include "runner.cuh"

#include <cassert>
#include <cstdlib>
#include <ctime>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <fmt/base.h>
#include <fmt/format.h>

#define CUDA_CHECK(err) sgemm::utils::cudaCheck(err, __FILE__, __LINE__)

namespace sgemm::utils {
void cudaCheck(hipError_t err, const char *file, int line) {
    if (err != hipSuccess) {
        fmt::println(
            "[CUDA ERROR] file={} line={}\n{}", file, line, hipGetErrorString(err)
        );
        exit(EXIT_FAILURE);
    }
}

void printCudaDeviceInfo() {
    int deviceId;
    CUDA_CHECK(hipGetDevice(&deviceId));
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, deviceId));
    // clang-format off
	fmt::println(
		R"(-------- DEVICE PROPERTIES --------
+ {:<21} : {}
+ {:<21} : {}
+ {:<21} : {}.{}
+ {:<21} : {}
+ {:<21} : {}
+ {:<21} : {}
+ {:<21} : {}
+ {:<21} : {}
+ {:<21} : {}
+ {:<21} : {}
+ {:<21} : {} B
+ {:<21} : {} GB
+ {:<21} : {} KB
+ {:<21} : {} KB
+ {:<21} : {} KB
+ {:<21} : {}
------------------------------------)",
		"deviceId", deviceId,
		"name", prop.name,
		"computeCapability", prop.major, prop.minor,
		"numSms", prop.multiProcessorCount,
		"maxThreadsPerBlock", prop.maxThreadsPerBlock,
		"maxThreadsPerSm", prop.maxThreadsPerMultiProcessor,
        "threadsPerWarp", prop.warpSize,
        "registersPerBlock", prop.regsPerBlock,
        "registersPerSm", prop.regsPerMultiprocessor,
        "numWarpsPerSm", prop.maxThreadsPerMultiProcessor / prop.warpSize,
        "memoryBusWidth", prop.memoryBusWidth,
		"totalGlobalMem", prop.totalGlobalMem / 1024 / 1024 / 1024,
		"sharedMemPerBlock", prop.sharedMemPerBlock / 1024,
		"sharedMemPerSm", prop.sharedMemPerMultiprocessor / 1024,
		"totalConstMem", prop.totalConstMem / 1024,
		"warpSize", prop.warpSize
	);
    // clang-format on
}

void l2Flush() {
    int deviceId;
    CUDA_CHECK(hipGetDevice(&deviceId));
    int l2CacheSize;
    CUDA_CHECK(hipDeviceGetAttribute(&l2CacheSize, hipDeviceAttributeL2CacheSize, deviceId));
    int *buf;  // we will use this to flush out CUDA L2 cache
    if (l2CacheSize > 0) {
        CUDA_CHECK(hipMalloc(&buf, l2CacheSize));
        CUDA_CHECK(hipMemsetAsync(buf, 0, l2CacheSize));  // flush the cache
        CUDA_CHECK(hipFree(buf));
    }
}

void runKernel(
    int kernelNum,
    const float *A,
    const float *B,
    float *C,
    float alpha,
    float beta,
    uint M,
    uint K,
    uint N,
    hipblasHandle_t handle
) {
    switch (kernelNum) {
        case 0: {
            // SGEMM cuBLAS reference in FP32.
            // NOTE: cuBLAS uses column-major order, but A, B, and C are row-major.
            // However, if we multiply B and A (in that order) in column-major order,
            // i.e., B.T@A.T = A@B, which is the same as multiplying A and B in
            // row-major order.
            // For reference, see: https://docs.nvidia.com/cuda/cublas/#cublasgemmex.
            hipblasGemmEx(
                handle,
                HIPBLAS_OP_N,  // don't transpose the first matrix (= B)
                HIPBLAS_OP_N,  // don't transpose the second matrix (= A)
                N,
                M,
                K,
                &alpha,      // alpha
                B,           // first matrix
                HIP_R_32F,  // fp32 (all matrices are fp32)
                // Stride to reach the next element in the first matrix; since our
                // matrices are row-major, we need to move a whole row of N elements to
                // get to the next element to make it seem like column-major.
                N,
                A,           // second matrix
                HIP_R_32F,  // fp32 (all matrices are fp32)
                // Stride to reach the next element in the second matrix; again, since
                // our matrices are row-major, we need to move a whole row of K elements
                // to get to the next element to make it seem like column-major.
                K,
                &beta,       // beta
                C,           // output matrix
                HIP_R_32F,  // fp32 (all matrices are fp32)
                // Stride to reach the next element in the output matrix; same as
                // before, move a whole row of N elements.
                N,
                HIPBLAS_COMPUTE_32F,            // internal compute precision, also fp32
                CUBLAS_GEMM_DEFAULT_TENSOR_OP  // use tensor core ops, if available
            );
            break;
        }
        case 1: {
            // Naive SGEMM kernel.
            dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 32));
            dim3 blockDim(32, 32);
            sgemm::kernels::sgemm_naive<<<gridDim, blockDim>>>(
                A, B, C, alpha, beta, M, K, N
            );
            break;
        }
        case 2: {
            // SGEMM kernel with coalesced global memory access.
            dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 32));
            dim3 blockDim(32, 32);
            sgemm::kernels::sgemm_gmem_coalesce<<<gridDim, blockDim>>>(
                A, B, C, alpha, beta, M, K, N
            );
            break;
        }
        case 3: {
            // SGEMM kernel with shared memory blocking.
            // NOTE: This kernel assumes square matrices; guards for non-square matrices
            // are not implemented.
            assert(M == N && N == K);
            // NOTE: This kernel assumes that M, N, and K are divisible by 32. There
            // are no explicit guardrails for this, so we assert it here.
            assert(M % 32 == 0 && N % 32 == 0 && K % 32 == 0);

            dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 32));
            dim3 blockDim(32, 32);
            // In this kernel, we don't use L1 cache (only SMEM is used). Hence, we
            // "carve out" all of the L1 to SMEM.
            hipFuncSetAttribute(reinterpret_cast<const void*>(
                sgemm::kernels::sgemm_smem_tiling<32>),
                hipFuncAttributePreferredSharedMemoryCarveout,
                cudaSharedmemCarveoutMaxShared
            );
            sgemm::kernels::sgemm_smem_tiling<32>
                <<<gridDim, blockDim>>>(A, B, C, alpha, beta, M, K, N);
            break;
        }
        default: {
            fmt::println("Kernel-{} not implemented", kernelNum);
            exit(EXIT_FAILURE);
        }
    }
}
}  // namespace sgemm::utils
